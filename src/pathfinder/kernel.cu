#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "../common/util.h"
#include "../common/App.h"

// Maze Values
//  0 -> Free position
//  1 -> Obstacle
//  2 -> Start Point
//  3 -> End Point
#define MAZE_FREE_POSITION 	0
#define MAZE_OBSTACLE 		1
#define MAZE_START_POINT	2
#define MAZE_END_POINT 		3

#define GetTransitionMatrix(Name, config) \
__global__ void \
Name(int *A, int *transitionMatrix) \
{\
	unsigned int width  = gridDim.x * blockDim.x;\
	unsigned int height = gridDim.y * blockDim.y;\
	unsigned int x      = blockIdx.x * blockDim.x + threadIdx.x;\
	unsigned int y      = blockIdx.y * blockDim.y + threadIdx.y;\
	unsigned int offset = x + y * width;\
\
	if (offset < (width*height))	{ \
		int nx = x;\
		int ny = y;\
		\
		config;\
		\
		int noffset = nx + ny * width;\
\
		if (noffset >= 0 && noffset < (width*height))\
		{\
			transitionMatrix[offset] = A[noffset] == MAZE_OBSTACLE? -1 : noffset;\
		}\
	}\
}


GetTransitionMatrix(Up, nx=x;ny=y+1;);
GetTransitionMatrix(Down, nx=x;ny=y-1;);
GetTransitionMatrix(Left, nx=x-1;ny=y;);
GetTransitionMatrix(Right, nx=x+1;ny=y;);
GetTransitionMatrix(UpRight, nx=x+1;ny=y+1;);
GetTransitionMatrix(UpLeft, nx=x-1;ny=y+1;);
GetTransitionMatrix(DownRight, nx=x+1;ny=y-1;);
GetTransitionMatrix(DownLeft, nx=x-1;ny=y-1;);

struct PathfindingData
{
	int *transitionUp;
	int *transitionDown;
	int *transitionLeft;
	int *transitionRight;
	int *transitionUpRight;
	int *transitionUpLeft;
	int *transitionDownRight;
	int *transitionDownLeft;
};

__device__  __host__ int getXY(int x, int y, int width)
{
	return y * width + x;
}



/**
 * Pathfinding using an iterative approach and Manhattan distance as heuristic value
 **/
__global__ void
GetPathUsingManhattanDistance(int *A, int *B, int startPointX, int startPointY, int goalPointX, int goalPointY)
{
	unsigned int width  = gridDim.x * blockDim.x;
	unsigned int height = gridDim.y * blockDim.y;
	unsigned int x      = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y      = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int offset = x + y * width;

	if (offset < (width*height))
	{
		int currentValue = A[offset];

		if (currentValue != MAZE_OBSTACLE)
		{
			currentValue = currentValue == MAZE_FREE_POSITION? 0 : 1;

			int mx, my;

			mx = __sad(x,startPointX,0);
			my = __sad(y,startPointY,0);
			int manhattan_from_start = max(mx,my);

			mx = __sad(x,goalPointX,0);
			my = __sad(y,goalPointY,0);
			int manhattan_from_goal = max(mx,my);

			//B[offset] = currentValue * (manhattan);
			B[offset] = manhattan_from_start + manhattan_from_goal;

			__syncthreads();

			A[offset] = B[offset];

			__syncthreads();

			int path_value = A[getXY(startPointX, startPointY, width)];

			B[offset] = A[offset] == path_value? manhattan_from_start : 0;
		}
		else
		{
			B[offset] = width*height;
		}
	}
}

void generateMaze(int *maze, int width, int height)
{
	int index = 0;
	int value = 0;

	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			// Generate walls
			if (i == 0 || i == height-1 || j == 0 || j == width-1)
			{
				value = MAZE_OBSTACLE;
			}
			else
			{
				value = MAZE_FREE_POSITION;
			}

			index = i * width + j;
			maze[index] = value;
		}
	}
}

// main routine
int main()
{
	const int WIDTH  = 32;
	const int HEIGHT = 32;
	const int SIZE 	 = WIDTH * HEIGHT;

	 // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, 0));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    if (sizeof(void *) != 8)
    {
        fprintf(stderr, "Unified Memory requires compiling for a 64-bit system.\n");
		hipDeviceReset();
#ifdef _WIN32
	USER_PAUSE;
#endif
		exit(EXIT_SUCCESS);
    }

    if (((deviceProp.major << 4) + deviceProp.minor) < 0x30)
    {
        fprintf(stderr, "requires Compute Capability of SM 3.0 or higher to run.\nexiting...\n");
        hipDeviceReset();
#ifdef _WIN32
	USER_PAUSE;
#endif
        exit(EXIT_SUCCESS);
    }

	//Reset no device
	CUDA_CHECK_RETURN(hipDeviceReset());

	int *maze, *maze_result;
	PathfindingData* pathfindingData;

	CUDA_CHECK_RETURN(hipMallocManaged((void**)&maze, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&maze_result, SIZE * sizeof(int)));

	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData, sizeof(PathfindingData)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionUp, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionLeft, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionRight, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionDown, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionUpLeft, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionUpRight, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionDownLeft, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&pathfindingData->transitionDownRight, SIZE * sizeof(int)));


	//Generate Maze
	generateMaze(maze, WIDTH, HEIGHT);

	// Copy Maze to Maze results matrix
	int index;
	for (int i = 0; i < HEIGHT; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			index = i * WIDTH + j;
			maze_result[index] = maze[index];
		}
	}

	// Set start point
	int START_POINT_X = 3,
		START_POINT_Y = 4;
	int START_POINT = getXY(START_POINT_X, START_POINT_Y, WIDTH);
	maze[START_POINT] = MAZE_START_POINT;
	// Set end point
	int END_POINT_X = 10,
		END_POINT_Y = 18;
	int END_POINT = getXY(END_POINT_X, END_POINT_Y, WIDTH);
	maze[END_POINT] = MAZE_END_POINT;

	printf("\nPathfinder - GPU\n");
	printf("Maze size: %d x %d - memory: [global]\n", WIDTH, HEIGHT);

#ifdef _WIN32
	USER_PAUSE;
#endif

	dim3 threadsPerBlock(32, 32);
	dim3 grid(WIDTH / threadsPerBlock.x, HEIGHT / threadsPerBlock.y);

	Up<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionUp);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	Down<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionDown);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	Right<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionRight);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	Left<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionLeft);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	UpRight<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionUpRight);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	UpLeft<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionUpLeft);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	DownRight<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionDownRight);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	DownLeft<<<grid, threadsPerBlock>>>(maze, pathfindingData->transitionDownLeft);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	/** Dummy approach - Get path using Manhattan Distance **/
	//
	//GetPathUsingManhattanDistance << <grid, threadsPerBlock >> > (maze, maze_result, START_POINT_X, START_POINT_Y, END_POINT_X, END_POINT_Y);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	// Showing results
	printf("\nPathfinder - GPU\n");
	printMatrix(pathfindingData->transitionUp, WIDTH, HEIGHT);
#ifdef _WIN32
	USER_PAUSE;
#endif

	// 5 - Free memory
	hipFree(maze);
	hipFree(maze_result);
	hipFree(pathfindingData->transitionDown);
	hipFree(pathfindingData->transitionDownLeft);
	hipFree(pathfindingData->transitionDownRight);
	hipFree(pathfindingData->transitionLeft);
	hipFree(pathfindingData->transitionRight);
	hipFree(pathfindingData->transitionUp);
	hipFree(pathfindingData->transitionUpLeft);
	hipFree(pathfindingData->transitionUpRight);
	hipFree(&pathfindingData);

	printf("End\n");
#ifdef _WIN32
	USER_PAUSE;
#endif

	return 0;
}

