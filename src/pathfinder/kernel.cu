#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "../common/util.h"
#include "../common/App.h"

// Maze Values
//  0 -> Free position
//  1 -> Obstacle
//  2 -> Start Point
//  3 -> End Point
#define MAZE_FREE_POSITION 	0
#define MAZE_OBSTACLE 		1
#define MAZE_START_POINT	2
#define MAZE_END_POINT 		3

__device__  __host__ int getXY(int x, int y, int width)
{
	return y * width + x;
}

/**
 * Pathfinding using an iterative approach and Manhattan distance as heuristic value
 **/
__global__ void
CalculateManhattanDistance(int *A, int *B, int goalPointX, int goalPointY)
{
	unsigned int width  = gridDim.x * blockDim.x;
	unsigned int height = gridDim.y * blockDim.y;
	unsigned int x      = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y      = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int offset = x + y * width;

	if (offset < (width*width))
	{
		int currentValue = A[offset];

		if (currentValue != MAZE_OBSTACLE)
		{
			currentValue = currentValue == MAZE_FREE_POSITION? 0 : 1;

			int manhattan = (x - goalPointX)*(x - goalPointX) + (y - goalPointY)*(y - goalPointY);

			//B[offset] = currentValue * (manhattan);
			B[offset] = manhattan;
		}
		else
		{
			B[offset] = width*height;
		}
	}
}


void generateMaze(int *maze, int width, int height)
{
	int index = 0;
	int value = 0;

	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			// Generate walls
			if (i == 0 || i == height-1 || j == 0 || j == width-1)
			{
				value = MAZE_OBSTACLE;
			}
			else
			{
				value = MAZE_FREE_POSITION;
			}

			index = i * width + j;
			maze[index] = value;
		}
	}
}

// main routine
int main()
{
	const int WIDTH  = 32;
	const int HEIGHT = 32;
	const int SIZE 	 = WIDTH * HEIGHT;

	 // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, 0));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    if (sizeof(void *) != 8)
    {
        fprintf(stderr, "Unified Memory requires compiling for a 64-bit system.\n");
		hipDeviceReset();
#ifdef _WIN32
	USER_PAUSE;
#endif
		exit(EXIT_SUCCESS);
    }

    if (((deviceProp.major << 4) + deviceProp.minor) < 0x30)
    {
        fprintf(stderr, "requires Compute Capability of SM 3.0 or higher to run.\nexiting...\n");
        hipDeviceReset();
#ifdef _WIN32
	USER_PAUSE;
#endif
        exit(EXIT_SUCCESS);
    }

	//Reset no device
	CUDA_CHECK_RETURN(hipDeviceReset());

	int *maze, *maze_result, *startPointX, *startPointY;

	CUDA_CHECK_RETURN(hipMallocManaged((void**)&maze, SIZE * sizeof(int)));
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&maze_result, SIZE * sizeof(int)));
	//CUDA_CHECK_RETURN(hipMallocManaged((void**)&startPointX, sizeof(int)));
	//CUDA_CHECK_RETURN(hipMallocManaged((void**)&startPointY, sizeof(int)));

	//Generate Maze
	generateMaze(maze, WIDTH, HEIGHT);

	// Copy Maze to Maze results matrix
	int index;
	for (int i = 0; i < HEIGHT; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			index = i * WIDTH + j;
			maze_result[index] = maze[index];
		}
	}

	// Set start point
	int START_POINT_X = 3,
		START_POINT_Y = 4;
	int START_POINT = getXY(START_POINT_X, START_POINT_Y, WIDTH);
	maze[START_POINT] = MAZE_START_POINT;
	// Set end point
	int END_POINT_X = 30,
		END_POINT_Y = 28;
	int END_POINT = getXY(END_POINT_X, END_POINT_Y, WIDTH);
	maze[END_POINT] = MAZE_END_POINT;

	printf("\nPathfinder - GPU\n");
	printf("Maze size: %d x %d - memory: [global]\n", WIDTH, HEIGHT);

#ifdef _WIN32
	USER_PAUSE;
#endif

	// Call Kernel
	dim3 threadsPerBlock(32, 32);
	dim3 grid(WIDTH / threadsPerBlock.x, HEIGHT / threadsPerBlock.y);

	CalculateManhattanDistance << <grid, threadsPerBlock >> > (maze, maze_result, END_POINT_X, END_POINT_Y);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	// Showing results
	printf("\nPathfinder - GPU\n");
	printMatrix(maze_result, WIDTH, HEIGHT);
#ifdef _WIN32
	USER_PAUSE;
#endif

	// 5 - Free memory
	hipFree(maze);
	hipFree(maze_result);

	printf("End\n");
#ifdef _WIN32
	USER_PAUSE;
#endif

	return 0;
}

